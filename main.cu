#include "hip/hip_runtime.h"
#include "main.cuh"

int main()
{
    // Create an array of points
    point *points = (point *)malloc(sizeof(point) * ITERATIONS_TO_DO);

    long int pointsWithinDistanceOfOneToOrigin = 0;

    long double x = 0;
    long double y = 0;

    for (long int i = 0; i < ITERATIONS_TO_DO_SQRT; i++)
    {
        for (long int j = 0; j < ITERATIONS_TO_DO_SQRT; j++)
        {
            points[(i * ITERATIONS_TO_DO_SQRT) + j].x = x;
            points[(i * ITERATIONS_TO_DO_SQRT) + j].y = y;
            x += MOVE_INCREMENT;
        }
        x = 0;
        y += MOVE_INCREMENT;
    }

    // and run the algorithm
    for (long int i = 0; i < ITERATIONS_TO_DO; i++)
    {
        if (std::abs(distance(0, points[i].x, 0, points[i].y)) < 1)
        {
            pointsWithinDistanceOfOneToOrigin++;
        }
    }

    // ITERATIONS_TO_DO needs to be made into a long double to force proper division. Otherwise, it will do integer division.
    long double pi = 4 * (pointsWithinDistanceOfOneToOrigin / (long double)ITERATIONS_TO_DO);

    // print the result
    printf("Pi is %Lf\n", pi);

    // free the allocated memory
    free(points);

    return 0;
}
