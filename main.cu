
#include <hip/hip_runtime.h>
#include <iostream>

#define iterationsToDo 4096
//#define iterationsToDo 25
#define isNumberEven(n) (n % 2 == 0)

__global__ void calculate(double *runningSum) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < iterationsToDo; i += stride) {

        double numerator = isNumberEven(i) ? 1 : -1;
        double denominator = 2 * i + 1;

        *runningSum += numerator / denominator;
    }
}

__global__ void init(double* runningSum) {
    *runningSum = 0;
}

__global__ void printTheThing(double *thingToPrint) {
    printf("!!! TEST !!!%.20lf", *thingToPrint * 4);
}

void doIt() {
    double *runningSum;

    if (hipMalloc((void **) &runningSum, sizeof(double)) != hipSuccess) {
        puts("Something truly terrible happened");
        exit(1);
    }
    init<<<1, 1>>>(runningSum);

    calculate<<<1, 1>>>(runningSum);
    hipDeviceSynchronize();

    printTheThing<<<1, 1>>>(runningSum);
    hipDeviceSynchronize();
}

int main() {
    doIt();
    return 0;
}
